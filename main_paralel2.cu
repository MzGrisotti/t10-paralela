#include "hip/hip_runtime.h"



#include <math.h>
#include <stdio.h>
#include <stdlib.h>


#include "Cube_Unity.h"
#include "bmp.h"

#define PI 3.1415

//Cube_Unity ***Cube;
//Cube_Unity *Cube;
//Cube_Unity ***Cube_Perspective;
//Cube_Unity *Cube_Perspective;

//funcao chamada continuamente. Deve-se controlar o que desenhar por meio de variaveis
//globais que podem ser setadas pelo metodo keyboard()
/*
void OffSet_Z(double offset){

   for(auto i = 0; i < Size; i++){
      for(auto j = 0; j < Size; j++){
         for(auto k = 0; k < Size; k++){
            Cube[i + Size * (j + Size * k)].z += offset;
         }
      }
   }

}

void Transform_3D_2D(){

   double dist_to_screen = 300;
   double new_x, new_y;

   for(auto i = 0; i < Size; i++){
      for(auto j = 0; j < Size; j++){
         for(auto k = 0; k < Size; k++){
            new_x = Cube[i + Size * (j + Size * k)].x*dist_to_screen/Cube[i + Size * (j + Size * k)].z+width/2;
            new_y = Cube[i + Size * (j + Size * k)].y*dist_to_screen/Cube[i + Size * (j + Size * k)].z+width/2;
            //Cube_Perspective[i][j][k].set_pos(new_x, new_y, 0, Cube[i + Size * (j + Size * k)].cor);
            Cube_Perspective[i + Size * (j + Size * k)].set_pos(new_x, new_y, 0, Cube[i + Size * (j + Size * k)].cor);
         }
      }
   }

}

void Rotate_x(double angulo){

   angulo = angulo * PI/180;

   double matrix_x[4][4] = {1,           0,            0, 0,
                           0, cos(angulo), -sin(angulo), 0,
                           0, sin(angulo),  cos(angulo), 0,
                           0,           0,            0, 1};

   double new_x, new_y, new_z;

   for(auto i = 0; i < Size; i++){
      for(auto j = 0; j < Size; j++){
         for(auto k = 0; k < Size; k++){

            new_x = Cube[i + Size * (j + Size * k)].x * matrix_x[0][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[0][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[0][2];
            new_y = Cube[i + Size * (j + Size * k)].x * matrix_x[1][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[1][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[1][2];
            new_z = Cube[i + Size * (j + Size * k)].x * matrix_x[2][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[2][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[2][2];

            Cube[i + Size * (j + Size * k)].x = new_x;
            Cube[i + Size * (j + Size * k)].y = new_y;
            Cube[i + Size * (j + Size * k)].z = new_z;

         }
      }
   }

}

void Rotate_y(double angulo){

   angulo = angulo * PI/180;

   double matrix_x[4][4] = { cos(angulo), 0, sin(angulo), 0,
                                      0, 1,           0, 0,
                           -sin(angulo), 0, cos(angulo), 0,
                                      0, 0,           0, 1};

   double new_x, new_y, new_z;

   for(auto i = 0; i < Size; i++){
      for(auto j = 0; j < Size; j++){
         for(auto k = 0; k < Size; k++){

            new_x = Cube[i + Size * (j + Size * k)].x * matrix_x[0][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[0][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[0][2];
            new_y = Cube[i + Size * (j + Size * k)].x * matrix_x[1][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[1][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[1][2];
            new_z = Cube[i + Size * (j + Size * k)].x * matrix_x[2][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[2][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[2][2];

            Cube[i + Size * (j + Size * k)].x = new_x;
            Cube[i + Size * (j + Size * k)].y = new_y;
            Cube[i + Size * (j + Size * k)].z = new_z;

         }
      }
   }

}

void Rotate_z(double angulo){

   angulo = angulo * PI/180;

   double matrix_x[4][4] = {cos(angulo), -sin(angulo), 0, 0,
                           sin(angulo),  cos(angulo), 0, 0,
                                     0,            0, 1, 0,
                                     0,            0, 0, 1};

   double new_x, new_y, new_z;

   for(auto i = 0; i < Size; i++){
      for(auto j = 0; j < Size; j++){
         for(auto k = 0; k < Size; k++){

            new_x = Cube[i + Size * (j + Size * k)].x * matrix_x[0][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[0][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[0][2];
            new_y = Cube[i + Size * (j + Size * k)].x * matrix_x[1][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[1][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[1][2];
            new_z = Cube[i + Size * (j + Size * k)].x * matrix_x[2][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[2][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[2][2];

            Cube[i + Size * (j + Size * k)].x = new_x;
            Cube[i + Size * (j + Size * k)].y = new_y;
            Cube[i + Size * (j + Size * k)].z = new_z;

         }
      }
   }

}
*/

void render(Cube_Unity *Cube, Cube_Unity *Cube_Perspective){
	int Size = 100;
	int width = 2000;
	int OffSet = 3000;
   double startAngulo = 1;
   //Rotate_z(angulo);
   double offset = startAngulo;
   
   auto k = blockIdx.x;
   int i = int(floor(threadIdx.x/10));
   
	//for(auto i = 0; i < Size; i++){
      //for(auto j = 0; j < Size; j++){
	  for(auto j = (threadIdx.x*10)%100; j < ((threadIdx.x*10)%100)+10; j++){
         //for(auto k = 0; k < Size; k++){
            Cube[i + Size * (j + Size * k)].z += offset;
         //}
      }
   //}
   
   //Rotate_y(angulo);
	double angulo = startAngulo * PI/180;

   double matrix_x[4][4] = { cos(angulo), 0, sin(angulo), 0,
                                      0, 1,           0, 0,
                           -sin(angulo), 0, cos(angulo), 0,
                                      0, 0,           0, 1};

   double new_x, new_y, new_z;

   //for(auto i = 0; i < Size; i++){
      for(auto j = (threadIdx.x*10)%100; j < ((threadIdx.x*10)%100)+10; j++){
         //for(auto k = 0; k < Size; k++){

            new_x = Cube[i + Size * (j + Size * k)].x * matrix_x[0][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[0][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[0][2];
            new_y = Cube[i + Size * (j + Size * k)].x * matrix_x[1][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[1][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[1][2];
            new_z = Cube[i + Size * (j + Size * k)].x * matrix_x[2][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[2][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[2][2];

            Cube[i + Size * (j + Size * k)].x = new_x;
            Cube[i + Size * (j + Size * k)].y = new_y;
            Cube[i + Size * (j + Size * k)].z = new_z;

         //}
      }
   //}
   
   //Rotate_x(angulo);
	angulo = startAngulo * PI/180;

   double matrix_y[4][4] = {1,           0,            0, 0,
                           0, cos(angulo), -sin(angulo), 0,
                           0, sin(angulo),  cos(angulo), 0,
                           0,           0,            0, 1};

   //double new_x, new_y, new_z;

   //for(auto i = 0; i < Size; i++){
      //for(auto j = 0; j < Size; j++){
	  for(auto j = (threadIdx.x*10)%100; j < ((threadIdx.x*10)%100)+10; j++){
         //for(auto k = 0; k < Size; k++){

            new_x = Cube[i + Size * (j + Size * k)].x * matrix_y[0][0] + Cube[i + Size * (j + Size * k)].y * matrix_y[0][1] + Cube[i + Size * (j + Size * k)].z * matrix_y[0][2];
            new_y = Cube[i + Size * (j + Size * k)].x * matrix_y[1][0] + Cube[i + Size * (j + Size * k)].y * matrix_y[1][1] + Cube[i + Size * (j + Size * k)].z * matrix_y[1][2];
            new_z = Cube[i + Size * (j + Size * k)].x * matrix_y[2][0] + Cube[i + Size * (j + Size * k)].y * matrix_y[2][1] + Cube[i + Size * (j + Size * k)].z * matrix_y[2][2];

            Cube[i + Size * (j + Size * k)].x = new_x;
            Cube[i + Size * (j + Size * k)].y = new_y;
            Cube[i + Size * (j + Size * k)].z = new_z;

         //}
      }
   //}
   //OffSet_Z(OffSet);
   offset = OffSet;
  //for(auto i = 0; i < Size; i++){
      //for(auto j = 0; j < Size; j++){
	  for(auto j = (threadIdx.x*10)%100; j < ((threadIdx.x*10)%100)+10; j++){
         //for(auto k = 0; k < Size; k++){
            Cube[i + Size * (j + Size * k)].z += offset;
         //}
      }
   //}
   //Transform_3D_2D();
   
   double dist_to_screen = 300;
   //double new_x, new_y;

   //for(auto i = 0; i < Size; i++){
      //for(auto j = 0; j < Size; j++){
	  for(auto j = (threadIdx.x*10)%100; j < ((threadIdx.x*10)%100)+10; j++){
         //for(auto k = 0; k < Size; k++){
            new_x = Cube[i + Size * (j + Size * k)].x*dist_to_screen/Cube[i + Size * (j + Size * k)].z+width/2;
            new_y = Cube[i + Size * (j + Size * k)].y*dist_to_screen/Cube[i + Size * (j + Size * k)].z+width/2;
            //Cube_Perspective[i][j][k].set_pos(new_x, new_y, 0, Cube[i + Size * (j + Size * k)].cor);
            Cube_Perspective[i + Size * (j + Size * k)].set_pos(new_x, new_y, 0, Cube[i + Size * (j + Size * k)].cor);
         //}
      }
   //}
   //OffSet_Z(-OffSet);
   offset = -OffSet;
   //for(auto i = 0; i < Size; i++){
      //for(auto j = 0; j < Size; j++){
	  for(auto j = (threadIdx.x*10)%100; j < ((threadIdx.x*10)%100)+10; j++){
         //for(auto k = 0; k < Size; k++){
            Cube[i + Size * (j + Size * k)].z += offset;
         //}
      }
   //}
}


int main(void)
{

	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	int maxThreadsPerBlock = devProp.maxThreadsPerBlock;
	int maxBlocks = devProp.maxGridSize[0];
	std::cout << "max threads per block: " << maxThreadsPerBlock << std::endl << "max blocks: " << maxBlocks << std::endl;

	Cube_Unity *Cube;
	Cube_Unity *Cube_Perspective;
	
	int Size = 100;
	int width = 2000;
	int OffSet = 3000;
	int frames = 1;
	double spacing_factor = 40;

   double inix, iniy, iniz;
   double atualx, atualy, atualz;
   double spacing;

   inix = iniy = iniz = atualx = atualy = atualz = -double(Size)/2 * spacing_factor;
   spacing = -inix*2 / double(Size);

   //Cube = new Cube_Unity**[Size];
   
   //Cube = new Cube_Unity[Size*Size*Size];
   hipMallocManaged(&Cube, Size*Size*Size*sizeof(Cube_Unity));
   //Cube_Perspective = new Cube_Unity[Size*Size*Size];
   hipMallocManaged(&Cube_Perspective, Size*Size*Size*sizeof(Cube_Unity));
   //Cube_Perspective = new Cube_Unity**[Size];
   for(auto i = 0; i < Size; i++){
       //Cube[i] = new Cube_Unity*[Size];
       //Cube_Perspective[i] = new Cube_Unity*[Size];
       for(auto j = 0; j < Size; j++){
           //Cube[i][j] = new Cube_Unity[Size];
           //Cube_Perspective[i][j] = new Cube_Unity[Size];
           for(auto k = 0; k < Size; k++){
              Cube[i + Size * (j + Size * k)].set_pos(atualx,atualy,atualz, 255);
              atualz += spacing;
           }
           atualz = iniz;
           atualy += spacing;
       }
       atualy = iniy;
       atualx += spacing;
   }

   // allocate picture array
   unsigned char* pic = new unsigned char[frames * width * width];

   for (int frame = 0; frame < frames; frame++) {
      for (int row = 0; row < width; row++) {
         for (int col = 0; col < width; col++) {
            unsigned char color = (unsigned char) 255;
            pic[frame * width * width + row * width + col] = (unsigned char) color;
         }
      }
   }

	int max_threads, max_blocks;

	max_threads = 1000;
	max_blocks = Size;
	
   for(int frame = 0; frame < frames; frame++){
	  render<<<max_blocks, max_threads>>>(Cube, Cube_Perspective);
      //render();
	  hipDeviceSynchronize();
      for(auto i = 0; i < Size; i++){
         for(auto j = 0; j < Size; j++){
            for(auto k = 0; k < Size; k++){
               //int row = (int)Cube_Perspective[i][j][k].x;
               //int col = (int)Cube_Perspective[i][j][k].y;
               int row = (int)Cube_Perspective[i + Size * (j + Size * k)].x;
               int col = (int)Cube_Perspective[i + Size * (j + Size * k)].y;
               if(row >= 0 && row < width && col >= 0 && col < width){
                  unsigned char color = (unsigned char) 0;
                  pic[frame * width * width + row * width + col] = (unsigned char) color;
               }
            }
         }
      }
   }

    for (int frame = 0; frame < frames; frame++) {
      char name[32];
      sprintf(name, "cube%d.bmp", frame + 1000);
      writeBMP(width, width, &pic[frame * width * width], name);
    }


}
